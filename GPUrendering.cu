#include "hip/hip_runtime.h"
// To build this code, use
// +-------------------------------------------------------------+
// | nvcc -ccbin=g++-4.4 -arch=sm_13 GPUrendering.cu -lGL -lglut|
// +-------------------------------------------------------------+

// CUDA - OpenGL interoperability

#define GL_GLEXT_PROTOTYPES
#include <cuda_gl_interop.h>

// OpenGL specific headers

#include <GL/gl.h>
#include <GL/glu.h>
#include <GL/glut.h>

// the usual gang of C++ headers

#include <iostream>
#include <complex>
#include <cmath>
#include <cstdlib>

// window size for displaying graphics

#define WIDTH  1024
#define HEIGHT 1024

// define a class for complex numbers and their operations

class dcmplx
{
public:
    double re;   // real component
    double im;   // imaginary component

// function to calculate the magnitude or absolute value of the complex number
// this function is called from and executes on the device (GPU) 

__device__
double magnitude()
{
    return pow((re*re + im*im),0.5);
}

};

// kernel to check all points inside the specified window for membership in the set
// and calculate an appropriate pixel colors for each point

__global__ void calculateMandelbrot(double xmin,
                                    double xmax,
                                    double ymin,
                                    double ymax,
                                    uchar4* ptr,
                                    const int MAX_ITER)
{
    double dx = (xmax - xmin)/WIDTH;  // grid spacing along X
    double dy = (ymax - ymin)/HEIGHT; // grid spacing along Y

    // global (i,j) location handled by this thread
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    int j = blockDim.y*blockIdx.y + threadIdx.y;

    // out-of-bounds threads return without doing enything
    if ((i >= WIDTH) || (j >= HEIGHT)) return;

    // offset using row-major ordering
    int offset = i + WIDTH*j;

    // calculate (x,y) potition
    double x = xmin + (double) i*dx;   // actual x coordinate (real component)
    double y = ymin + (double) j*dy;   // actual y coordinate (imaginary component)

    // carry out the iterative check
    // z <---- z*z + c
    dcmplx c;
    c.re = x;
    c.im = y;

    dcmplx z;
    z.re = 0.0;
    z.im = 0.0;

    int iter = 0;

    while(iter<MAX_ITER)
    {
        iter++;
        dcmplx temp = z;
        z.re = temp.re*temp.re - temp.im*temp.im  +  c.re;
        z.im = 2.0*temp.re*temp.im                +  c.im;
        
        if (z.magnitude() > 2.0) break;
    }

    // "iter" now stores how many iterations were required for divergence
    // for points outside the Mandelbrot set, this is typically a small number
    // points inside the set do not diverge and thus iter is a large number for such points

    // assign pixel color based on the number of iterations - Red Green Blue (RGB) 

    float R, G, B;

    if(iter==MAX_ITER)
    {
        // this point is inside the Mandelbrot set. Paint it black.
        R = 0;
        G = 0;
        B = 0;
    }
    else
    {
        // ratio of iterations required to escape
        // the higher this value, the closer the point is to the set
        float frac = (float) iter / MAX_ITER;

        if(frac<=0.5)
        {
            // yellow to blue transition
            R = 2*frac;
            G = 2*frac;
            B = 1 - 2*frac;
        }
        else
        {
            // red to yellow transition
            R = 1;
            G = 2 - 2*frac;
            B = 0;
        }
    }

    // convert pixel color from float(0-1) to int(0-255)
    // (unsigned char is an eight bit integer)
    //
    // 0000 0000      0
    // 0000 0001      1
    // 0000 0010      2
    // 0000 0011      3
    // 0000 0100      4
    //     .          .
    //     .          .
    //     .          .
    // 1111 1111    255

    ptr[offset].x = (int) 255*R;
    ptr[offset].y = (int) 255*G;
    ptr[offset].z = (int) 255*B;
}

// minimum and maximum X and Y coordinates

void showMandelbrot(double xmin, double xmax, double ymin, double ymax)
{
    //----------------------------------------------------------------
    //  Use GPU for calculating the 2D array of "iterations to escape"
    //----------------------------------------------------------------

    // problem parameters
    const int MAX_ITER = 200;

    // create variables that will be shared between OpenGL and CUDA device
    GLuint bufferObj;
    hipGraphicsResource *resource;

    hipDeviceProp_t prop;
    int device;

    memset(&prop, 0, sizeof(hipDeviceProp_t));

    // choose a CUDA capable device that is at least compute 1.3
    prop.major = 1;
    prop.minor = 3;
    hipChooseDevice( &device, &prop);

    // tell the runtime that we intend to use this device for CUDA and OpenGL
    cudaGLSetGLDevice(device);

    // generate a pixel buffer object (PBO)
    glGenBuffers(1, &bufferObj);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, WIDTH * HEIGHT * 4, NULL, GL_DYNAMIC_DRAW_ARB);

    // register "bufferObj" with the CUDA runtime as a graphics resource
    hipGraphicsGLRegisterBuffer(&resource, bufferObj, cudaGraphicsMapFlagsNone);

    // create a pointer in device memory for CUDA to use the buffer object
    uchar4* devPtr;
    size_t size;

    hipGraphicsMapResources(1, &resource, NULL);
    hipGraphicsResourceGetMappedPointer( (void**) &devPtr, &size, resource);

    // launch CUDA kernel to populate the device buffer
    const int tx = 32;
    const int ty = 32;

    dim3 threads(tx, ty, 1);

    const int bx = ceil( (float) WIDTH  / (float) tx );
    const int by = ceil( (float) HEIGHT / (float) ty );

    dim3 blocks( bx, by, 1);

    calculateMandelbrot<<<blocks,threads>>>(xmin, xmax, ymin, ymax, devPtr, MAX_ITER);

    // make sure CUDA kernel is finished before plotting the results
    hipGraphicsUnmapResources(1, &resource, NULL);

    //--------------------------------
    //  Render the image using OpenGL
    //--------------------------------

    // select background color to be white
    // R = 1, G = 1, B = 1, alpha = 0
    glClearColor (1.0, 1.0, 1.0, 0.0);
  
    // initialize viewing values
    glMatrixMode(GL_PROJECTION);
  
    // replace current matrix with the identity matrix
    glLoadIdentity();
  
    // set clipping planes in the X-Y-Z coordinate system
    glOrtho(xmin,xmax,ymin,ymax, -1.0, 1.0);
  
    // clear all pixels
    glClear (GL_COLOR_BUFFER_BIT);

    // render pixel data from buffer already in GPU memory
    glDrawPixels(WIDTH, HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);

    // divide rendered image into 4 quads separated by thick white lines
    glColor3f(255,255,255);
    double dx = (xmax - xmin)/WIDTH;  // grid spacing along X
    double dy = (ymax - ymin)/HEIGHT; // grid spacing along Y
    glRectf(xmin+(xmax-xmin)/2-1.95*dx,ymin,xmin+(xmax-xmin)/2+1.95*dx,ymax);
    glRectf(xmin,ymin+(ymax-ymin)/2-1.95*dy,xmax,ymin+(ymax-ymin)/2+1.95*dy);

    glFlush ();
}

// Entry point for the display routine

void display(void)
{
    // specify initial window size in the X-Y plane
    double xmin = -2, xmax = 1, ymin = -1.5, ymax = 1.5;

    // user selection of appropriate quadrant
    int choice;  

    std::cout << " +------+------+ " << std::endl;
    std::cout << " |      |      | " << std::endl;
    std::cout << " |  1   |   2  | " << std::endl;
    std::cout << " |      |      | " << std::endl;
    std::cout << " +------+------+ " << std::endl;
    std::cout << " |      |      | " << std::endl;
    std::cout << " |  3   |   4  | " << std::endl;
    std::cout << " |      |      | " << std::endl;
    std::cout << " +------+------+ " << std::endl;

    // infinite loop until user kills this process
    while(true)
    {
        // display the Mandelbrot set in (xmin,ymin)-(xmax,ymax)
        showMandelbrot(xmin,xmax,ymin,ymax);

        // ask user for selecting a region for further zoom-in
        std::cout << "Zoom in to <1,2,3,4>: ";
        std::cin >> choice;

        // update display limits based on user choice
        switch (choice) 
        {
            case 1:
                xmax = xmin + (xmax - xmin)/2; 
                ymin = ymin + (ymax - ymin)/2;
                break;
            case 2:
                xmin = xmin + (xmax - xmin)/2; 
                ymin = ymin + (ymax - ymin)/2;
                break;
            case 3:
                xmax = xmin + (xmax - xmin)/2; 
                ymax = ymin + (ymax - ymin)/2;
                break;
            case 4:
                xmin = xmin + (xmax - xmin)/2; 
                ymax = ymin + (ymax - ymin)/2;
                break;
        }
    }
}

int main(int argc, char* argv[])
{
    //--------------------------------
    //   Create a WINDOW using GLUT
    //--------------------------------

    // launch the GLUT runtime
    glutInit(&argc, argv);

    // set the window's display mode
    glutInitDisplayMode (GLUT_SINGLE | GLUT_RGB);

    // set the windows width and height
    glutInitWindowSize (WIDTH, HEIGHT);

    // location of top left corner of window
    glutInitWindowPosition (20, 0);      

    // create a window with the specified title
    glutCreateWindow ("Mandelbrot Set");

    //---------------------------------------------
    // Display something in the window using OpenGL
    //---------------------------------------------

    // pass a function pointer
    glutDisplayFunc(display);

    // GLUT processing loop continues until the application terminates
    glutMainLoop();

    return 0;
}
